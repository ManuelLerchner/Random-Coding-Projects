
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
using namespace std;
#include <fstream>
#include <cmath>


//nvcc .\raytracing.cu -o raytracing
//.\raytracing

// limited version of checkCudaErrors from helper_cuda.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;

    float u=float(i) / max_x;
    float v=float(j) / max_y;
    fb[pixel_index + 0] = u;
    fb[pixel_index + 1] = v;
    fb[pixel_index + 2] = 0.2;

    
    if(sqrt(pow(u-0.5,2)+pow(v-0.5,2))<0.25){
        fb[pixel_index + 0] = 1;
        fb[pixel_index + 1] = 0;
        fb[pixel_index + 2] = 0;
    }


}

int main() {
    int nx = 720;
    int ny = 720;
    int tx = 128;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);

    // allocate FB
    float *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    std::cerr << "begin saving the file.\n";

    ofstream ofs("first.ppm", ios_base::out | ios_base::binary);
    ofs << "P3" << endl << nx << ' ' << ny << endl << "255" << endl;

        for (int j = ny-1; j >= 0; j--) {
            for (int i = 0; i < nx; i++) {
                size_t pixel_index = j*3*nx + i*3;
                float r = fb[pixel_index + 0];
                float g = fb[pixel_index + 1];
                float b = fb[pixel_index + 2];
                int ir = int(255.99*r);
                int ig = int(255.99*g);
                int ib = int(255.99*b);
                ofs << ir << " " << ig << " " << ib << "\n";
            }
        }
    ofs.close();
    std::cerr << "finished!\n";

    checkCudaErrors(hipFree(fb));
}

